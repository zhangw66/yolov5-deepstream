#include "hip/hip_runtime.h"
#include <assert.h>
#include "yololayer.h"
#include "utils.h"

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, const std::vector<Yolo::YoloKernel>& vYoloKernel)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        mClassCount = classCount;
        mYoloV5NetWidth = netWidth;
        mYoloV5NetHeight = netHeight;
        mMaxOutObject = maxOut;
        mYoloKernel = vYoloKernel;
        mKernelCount = vYoloKernel.size();
	std::cout << "class count: " << mClassCount
		  << " NET WIDTH: " << mYoloV5NetWidth
		  << " NET HEIGHT: " << mYoloV5NetHeight
		  << " max out: " << mMaxOutObject
		  << " thread count: " << mThreadCount
		  << " kernel count: " << mKernelCount << std::endl;
        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
	    std::cout << "width: " << yolo.width;
	    std::cout << " height: " << yolo.height;
	    for (int iii = 0; iii < CHECK_COUNT * 2;iii++)
	    std::cout << " anchors: " << yolo.anchors[iii] << std::endl;
	    
	    CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    YoloLayerPlugin::~YoloLayerPlugin()
    {
	    printf("%s:%d\n", __func__, __LINE__);
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipFree(mAnchor[ii]));
        }
        CUDA_CHECK(hipHostFree(mAnchor));
    }

    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        read(d, mYoloV5NetWidth);
        read(d, mYoloV5NetHeight);
        read(d, mMaxOutObject);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(mYoloKernel.data(), d, kernelSize);
        d += kernelSize;
        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer) const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        write(d, mYoloV5NetWidth);
        write(d, mYoloV5NetHeight);
        write(d, mMaxOutObject);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(d, mYoloKernel.data(), kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }

    size_t YoloLayerPlugin::getSerializationSize() const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size() + sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight) + sizeof(mMaxOutObject);
    }

    int YoloLayerPlugin::initialize()
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return 0;
    }

    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        //output the result to channel
        int totalsize = mMaxOutObject * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloLayerPlugin::getPluginNamespace() const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return false;
    }

    void YoloLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
	    printf("%s:%d\n", __func__, __LINE__);
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
	    printf("%s:%d\n", __func__, __LINE__);
    }

    // Detach the plugin object from its execution context.
    void YoloLayerPlugin::detachFromContext() {}

    const char* YoloLayerPlugin::getPluginType() const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return "YoloLayer_TRT";
    }

    const char* YoloLayerPlugin::getPluginVersion() const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return "1";
    }

    void YoloLayerPlugin::destroy()
    {
	    printf("%s:%d\n", __func__, __LINE__);
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloLayerPlugin::clone() const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        YoloLayerPlugin* p = new YoloLayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, mYoloKernel);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }
#if 1
    __device__ float Logist(float data) { return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output, int noElements,
        const int netwidth, const int netheight, int maxoutobject, int yoloWidth, int yoloHeight, const float anchors[CHECK_COUNT * 2], int classes, int outputElem)
    {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid * bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < 3; ++k) {
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (box_prob < IGNORE_THRESH) continue;
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float *res_count = output + bnIdx * outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= maxoutobject) return;
            char* data = (char *)res_count + sizeof(float) + count * sizeof(Detection);
            Detection* det = (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            // pytorch:
            //  y = x[i].sigmoid()
            //  y[..., 0:2] = (y[..., 0:2] * 2. - 0.5 + self.grid[i].to(x[i].device)) * self.stride[i]  # xy
            //  y[..., 2:4] = (y[..., 2:4] * 2) ** 2 * self.anchor_grid[i]  # wh 
            //  X: (sigmoid(tx) + cx)/FeaturemapW *  netwidth 
            det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * netwidth / yoloWidth;
            det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * netheight / yoloHeight;

            // W: (Pw * e^tw) / FeaturemapW * netwidth  
            // v5: https://github.com/ultralytics/yolov5/issues/471
            det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2 * k];
            det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2 * k + 1];
            det->conf = box_prob * max_cls_prob;
            det->class_id = class_id;
        }
    }
#endif

#if 0
    __device__ float Logist(float data){ return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes,int outputElem) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid*bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < 3; ++k) {
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (box_prob < IGNORE_THRESH) continue;
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float *res_count = output + bnIdx*outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= MAX_OUTPUT_BBOX_COUNT) return;
            char* data = (char *)res_count + sizeof(float) + count * sizeof(Detection);
            Detection* det =  (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * INPUT_W / yoloWidth;
            det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * INPUT_H / yoloHeight;
            det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2*k];
            det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2*k + 1];
            det->conf = box_prob * max_cls_prob;
            det->class_id = class_id;
        }
    }
#endif
    
    void YoloLayerPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx * outputElem, 0, sizeof(float)));
        }
        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height*batchSize; 
            if (numElem < mThreadCount)
                mThreadCount = numElem;

            //printf("Net: %d  %d \n", mYoloV5NetWidth, mYoloV5NetHeight);
#if 1
	    //CalDetection << < (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount >> >
     	    CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[i], output, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, yolo.width, yolo.height, (float *)mAnchor[i], mClassCount, outputElem);
#else
     	    CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[i], output, numElem, yolo.width, yolo.height, (float *)mAnchor[i], mClassCount, outputElem);
#endif
        }
    }
#if 0
    void YoloLayerPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {

        int outputElem = 1 + MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        for(int idx = 0 ; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx*outputElem, 0, sizeof(float)));
        }
        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height*batchSize;
            if (numElem < mThreadCount)
                mThreadCount = numElem;
            CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream>>>
                (inputs[i], output, numElem, yolo.width, yolo.height, (float *)mAnchor[i], mClassCount, outputElem);
        }

    }
    
#endif

    int YoloLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

    YoloPluginCreator::YoloPluginCreator()
    {
	    printf("%s:%d\n", __func__, __LINE__);
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames()
    {
	    printf("%s:%d\n", __func__, __LINE__);
        return &mFC;
    }

    IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        int class_count = 80;
        int input_w = 416;
        int input_h = 416;
        int max_output_object_count = 1000;
        std::vector<Yolo::YoloKernel> yolo_kernels(3);

        const PluginField* fields = fc->fields;
        for (int i = 0; i < fc->nbFields; i++) {
            if (strcmp(fields[i].name, "netdata") == 0) {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                int *tmp = (int*)(fields[i].data);
                class_count = tmp[0];
                input_w = tmp[1];
                input_h = tmp[2];
                max_output_object_count = tmp[3];
            } else if (strstr(fields[i].name, "yolodata") != NULL) {
                assert(fields[i].type == PluginFieldType::kFLOAT32);
                int *tmp = (int*)(fields[i].data);
                YoloKernel kernel;
                kernel.width = tmp[0];
                kernel.height = tmp[1];
                for (int j = 0; j < fields[i].length - 2; j++) {
                    kernel.anchors[j] = tmp[j + 2];
                }
                yolo_kernels[2 - (fields[i].name[8] - '1')] = kernel;
            }
        }
        YoloLayerPlugin* obj = new YoloLayerPlugin(class_count, input_w, input_h, max_output_object_count, yolo_kernels);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
	    printf("%s:%d\n", __func__, __LINE__);
        // This object will be deleted when the network is destroyed, which will
        // call YoloLayerPlugin::destroy()
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
}
